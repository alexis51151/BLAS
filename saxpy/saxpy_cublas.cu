#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cassert>
#include <vector>

#include "../cuda_error.h" // File with macros to handle CUDA errors

using namespace std;

/*
 * SAXPY stands for Single Precision A * X + Y where X,Y are vectors of size N.
 * Computation: Let a \in R, \forall i \in [1,N],
 * z_i = a * x_i + y_i
 */

__global__
void saxpy_kernel(int n, float a, const float* x, const float* y, float* z) {
    // 2D -> 1D index
    uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
    // saxpy computation
    if (id < n) {
        z[id] = a * x[id] + y[id];
    }
}

template<typename T>
void bench_cuda(int n_trials, T func_saxpy) {
    // Define the example: 2 * I + 3I
    const int N = 2 << 27;
    float a = 2.0;

    // Allocate the vectors on host
    float* x = (float*) malloc(N * sizeof(float));
    float* y = (float*) malloc(N * sizeof(float));
    float* z = (float*) malloc(N * sizeof(float));

    // Initialize the vectors on host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0;
        y[i] = 3.0;
        z[i] = 0.0;
    }

    // Allocate the vectors on device
    float* d_x;
    float* d_y;
    float* d_z;
    CHECK_CUDA_ERROR(hipMalloc(&d_x, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_y, N * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_z, N * sizeof(float)));

    // Copy the vectors from host to devices
    CHECK_CUDA_ERROR(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice));
    // No need to copy z to device, as its values will be overwritten

    // Compute the number of blocks
    int threads_per_block = 1024; // number of threads per block
    int n_blocks = (N + threads_per_block - 1) / threads_per_block; // ceil(N / threads_per_block)

    long avg_dur = 0;
    for (int i = 0; i < n_trials; i++) {
        auto start = chrono::high_resolution_clock::now();
        func_saxpy<<<n_blocks, threads_per_block>>>(N, a, d_x, d_y, d_z);
        CHECK_LAST_CUDA_ERROR();
        // Device synchronization barrier
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        auto end = chrono::high_resolution_clock::now();
        long dur_ms = chrono::duration_cast<chrono::milliseconds>(end - start).count();
        avg_dur += dur_ms;
    }
    avg_dur = avg_dur / n_trials;
    cout << "[saxpy_cuda]: avg(duration) = " << avg_dur << " ms" << endl;

    // Copy back the output from device to host
    CHECK_CUDA_ERROR(hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost));

    // Check the output for the last call
    double sum = 0.0;
    for (int i = 0; i < N; i++) {
        assert(z[i] == 5);
        sum += z[i];
    }
    cout << "sum = " << sum << endl;
    assert(sum == N * 5);

    // Free the vectors on device
    CHECK_CUDA_ERROR(hipFree(d_x));
    CHECK_CUDA_ERROR(hipFree(d_y));
    CHECK_CUDA_ERROR(hipFree(d_z));

    // Free the vectors on host
    free(x);
    free(y);
    free(z);
}


int main() {
    bench_cuda(10, saxpy_kernel);
    return 0;
}
